// local_engine/cuda/matrix_ops.cu

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA failure at line %d: %s\n", __LINE__, hipGetErrorString(status)); \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

template <typename T>
__global__ void matrix_multiply_kernel(
    const T* A, const T* B, T* C, 
    int M, int N, int K, 
    T alpha, T beta
) {
    // Block size and tile dimensions
    const int TILE_SIZE = 16;
    __shared__ T As[TILE_SIZE][TILE_SIZE];
    __shared__ T Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    T sum = 0.0;

    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load tiles into shared memory
        if (row < M && t*TILE_SIZE + threadIdx.x < K) {
            As[threadIdx.y][threadIdx.x] = A[row*K + t*TILE_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (t*TILE_SIZE + threadIdx.y < K && col < N) {
            Bs[threadIdx.y][threadIdx.x] = B[(t*TILE_SIZE + threadIdx.y)*N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        // Compute partial sum
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row*N + col] = alpha * sum + beta * C[row*N + col];
    }
}

__global__ void relu_activation_kernel(
    half* input, half* output, 
    int elements, half negative_slope
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elements) {
        float val = __half2float(input[idx]);
        output[idx] = __float2half(val > 0 ? val : val * __half2float(negative_slope));
    }
}

__global__ void gelu_activation_kernel(
    float* input, float* output, 
    int elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elements) {
        float x = input[idx];
        output[idx] = 0.5 * x * (1.0 + tanhf(0.7978845608 * (x + 0.044715 * x * x * x)));
    }
}

__global__ void matrix_transpose_kernel(
    const float* input, float* output,
    int rows, int cols
) {
    __shared__ float tile[32][32+1]; // +1 to avoid bank conflicts

    int x = blockIdx.x * 32 + threadIdx.x;
    int y = blockIdx.y * 32 + threadIdx.y;

    if (x < cols && y < rows) {
        tile[threadIdx.y][threadIdx.x] = input[y * cols + x];
    }

    __syncthreads();

    x = blockIdx.y * 32 + threadIdx.x;
    y = blockIdx.x * 32 + threadIdx.y;

    if (x < rows && y < cols) {
        output[y * rows + x] = tile[threadIdx.x][threadIdx.y];
    }
}

// Batch matrix multiplication (3D tensor support)
template <typename T>
__global__ void batch_matmul_kernel(
    const T* A, const T* B, T* C,
    int batch_size, int M, int N, int K,
    T alpha, T beta
) {
    extern __shared__ __align__(sizeof(T)) unsigned char shared_mem[];
    T* As = reinterpret_cast<T*>(shared_mem);
    T* Bs = As + blockDim.y * blockDim.z;

    int batch = blockIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    T sum = 0.0;

    for (int t = 0; t < (K + blockDim.z - 1) / blockDim.z; ++t) {
        // Load tiles from global to shared memory
        if (row < M && t*blockDim.z + threadIdx.z < K) {
            As[threadIdx.y * blockDim.z + threadIdx.z] = 
                A[batch*M*K + row*K + t*blockDim.z + threadIdx.z];
        } else {
            As[threadIdx.y * blockDim.z + threadIdx.z] = 0.0;
        }

        if (t*blockDim.z + threadIdx.y < K && col < N) {
            Bs[threadIdx.z * blockDim.y + threadIdx.y] = 
                B[batch*K*N + (t*blockDim.z + threadIdx.y)*N + col];
        } else {
            Bs[threadIdx.z * blockDim.y + threadIdx.y] = 0.0;
        }

        __syncthreads();

        // Compute partial sum
        for (int k = 0; k < blockDim.z; ++k) {
            sum += As[threadIdx.y * blockDim.z + k] 
                 * Bs[k * blockDim.y + threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[batch*M*N + row*N + col] = alpha * sum + beta * C[batch*M*N + row*N + col];
    }
}

// Rust FFI interface
extern "C" {

void cuda_matrix_multiply_f32(
    const float* A, const float* B, float* C,
    int M, int N, int K,
    float alpha, float beta,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid((N + 15)/16, (M + 15)/16);
    matrix_multiply_kernel<<<grid, block, 0, stream>>>(A, B, C, M, N, K, alpha, beta);
}

void cuda_matrix_multiply_f16(
    const half* A, const half* B, half* C,
    int M, int N, int K,
    half alpha, half beta,
    hipStream_t stream
) {
    dim3 block(16, 16);
    dim3 grid((N + 15)/16, (M + 15)/16);
    matrix_multiply_kernel<<<grid, block, 0, stream>>>(A, B, C, M, N, K, alpha, beta);
}

void cuda_batch_matmul_f32(
    const float* A, const float* B, float* C,
    int batch_size, int M, int N, int K,
    float alpha, float beta,
    hipStream_t stream
) {
    dim3 block(16, 16, 4);
    dim3 grid((N + 15)/16, (M + 15)/16, batch_size);
    size_t shared_mem_size = 2 * 16*16*4 * sizeof(float);
    batch_matmul_kernel<<<grid, block, shared_mem_size, stream>>>(
        A, B, C, batch_size, M, N, K, alpha, beta
    );
}

void cuda_relu_activation_f16(
    half* input, half* output,
    int elements, half negative_slope,
    hipStream_t stream
) {
    int block_size = 256;
    int grid_size = (elements + block_size - 1) / block_size;
    relu_activation_kernel<<<grid_size, block_size, 0, stream>>>(
        input, output, elements, negative_slope
    );
}

void cuda_gelu_activation_f32(
    float* input, float* output,
    int elements,
    hipStream_t stream
) {
    int block_size = 256;
    int grid_size = (elements + block_size - 1) / block_size;
    gelu_activation_kernel<<<grid_size, block_size, 0, stream>>>(
        input, output, elements
    );
}

void cuda_matrix_transpose_f32(
    const float* input, float* output,
    int rows, int cols,
    hipStream_t stream
) {
    dim3 block(32, 32);
    dim3 grid((cols + 31)/32, (rows + 31)/32);
    matrix_transpose_kernel<<<grid, block, 0, stream>>>(input, output, rows, cols);
}

} // extern "C"
